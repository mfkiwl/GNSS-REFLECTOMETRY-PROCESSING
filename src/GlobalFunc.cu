#include "hip/hip_runtime.h"
//==========================================================================
// Author: Oriol Cervelló (oriol.cn [at] protonmail.com) 
//==========================================================================
// License: GNU GPLv3.0
// Copyright (C) 2019  Oriol Cervelló
//
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
// 
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
// 
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.
//==========================================================================

#include "GlobalFunc.cuh"

//GLOBAL FUNCTIONS

__global__ void multip(int samples, hipfftComplex *data1, hipfftComplex *data2, int refsize, bool interferometric)
{
	hipfftComplex aux;
	int k;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < samples; i += stride) {
		if (interferometric == true) {
			k = i;
		}
		else {
			k = i % refsize;
		}
		

		//(a+bi)*(c+di)=(ac−bd)+(ad+bc)i

		aux.x = data1[i].x* data2[k].x - data1[i].y*(-data2[k].y);
		aux.y = data1[i].x*(-data2[k].y) + data1[i].y*data2[k].x;


		data1[i].x = aux.x;
		data1[i].y = aux.y;
	}
}

__global__ void extendRefSignal(int samples, hipfftComplex *data, int refsize) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < samples; i += stride) {

		if (i >= refsize) {
			data[i] = data[i%refsize];

		}
	}
}

__global__ void applyDoppler(int samples, hipfftComplex *data, float freqDoppler, float fs, unsigned long long int samplePhaseMantain,
	int origSamples, int ddmQuant, int ddmRes, int fftsize)
{
	hipfftComplex aux, aux2;
	double angle, freq, phasemantain;

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < samples; i += stride) {
		
		if (ddmQuant > 1) {
			phasemantain = ((i % (origSamples)));//origninal samples signal
			freq = freqDoppler - (ddmRes * (ddmQuant / 2)) + ((i / (origSamples))*(ddmRes));
		}
		else{ 
			freq = freqDoppler;
			phasemantain = i;//origninal samples signal
		}
		angle = 2.0*PI*double((phasemantain) + samplePhaseMantain)*(freq / double(fs));
		//angle = 2.0*PI*(phasemantain + 0)*(freq / fs);
		aux2.x = cos(angle);
		aux2.y = sin(angle);

		//(a+bi)*(c+di)=(ac−bd)+(ad+bc)i
		aux.x = data[i].x*aux2.x - data[i].y*aux2.y;
		aux.y = data[i].x*aux2.y + data[i].y*aux2.x;

		data[i].x = aux.x;
		data[i].y = aux.y;
	}
}

__global__ void selectMaxs(int numOfFFT, int quantOfIncohSumAve, int ddmQuant, int *arrayPos, Npp32f *deviceArrayMaxs) {

	int step = numOfFFT / quantOfIncohSumAve;

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < step; i += stride) {
		//if (i < step) {
		for (int j = i; j < step*ddmQuant - 1; j = j + step) {
			if (deviceArrayMaxs[j + step] > deviceArrayMaxs[i]) {
				deviceArrayMaxs[i] = deviceArrayMaxs[j + step];
				arrayPos[i] = arrayPos[j + step];

			}
		}
		//}
	}
}

__global__ void savePeak(int numOfFFT, hipfftComplex *dataFromIFFT, hipfftComplex *dataToSave, int peakSamplesToSave,
	int quantOfIncohSumAve, int fftsize, int *arrayPos, int ddmQuant) {

	int samplesToSave = numOfFFT * peakSamplesToSave*ddmQuant;
	int posOnIFFT, fftOfThePeak, indexOfArrayPos, leftPosMaxOnOneIFFT, posOnOneIFFT;//rightPosMaxOnOneIFFT;

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < samplesToSave; i += stride) {

		fftOfThePeak = i / peakSamplesToSave;//num of FFT in dataFromInv
		indexOfArrayPos = fftOfThePeak / quantOfIncohSumAve;//number of index in arrayPos
		//rightPosMaxOnOneIFFT = (arrayPos[indexOfPos] + peakSamplesToSave / 2);
		leftPosMaxOnOneIFFT = arrayPos[indexOfArrayPos % (numOfFFT / quantOfIncohSumAve)] - (peakSamplesToSave / 2);//begining of data to save
		posOnOneIFFT = leftPosMaxOnOneIFFT + (i%peakSamplesToSave);// sample of i in one fft

		if (posOnOneIFFT >= fftsize) {
			posOnIFFT = fftOfThePeak * fftsize + posOnOneIFFT % fftsize; //sample in the data from IFFT
			dataToSave[i] = dataFromIFFT[posOnIFFT];

			//case 2
		}
		else if (posOnOneIFFT < 0) {
			posOnIFFT = fftOfThePeak * fftsize + (fftsize + posOnOneIFFT);//sample in the data from IFFT
			dataToSave[i] = dataFromIFFT[posOnIFFT];
			//case 3
		}
		else {
			posOnIFFT = fftOfThePeak * fftsize + posOnOneIFFT;
			dataToSave[i] = dataFromIFFT[posOnIFFT];//sample in the data from IFFT
			//case 1
		}
	}
}

__global__ void inchoerentSum(int samplesInchoerentSum, hipfftComplex *dataFromInv, Npp32f *dataStorageInocherentSum,
	int quantofAverageIncoherent, int fftsize)
{

	int indexofInv, numofSumM;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < samplesInchoerentSum; i += stride) {
		dataStorageInocherentSum[i] = 0;
		numofSumM = i / fftsize;
		for (int k = 0; k < quantofAverageIncoherent; k++) {
			indexofInv = numofSumM * quantofAverageIncoherent*fftsize + k * fftsize + i % fftsize;
			dataStorageInocherentSum[i] += dataFromInv[indexofInv].x*dataFromInv[indexofInv].x + dataFromInv[indexofInv].y*dataFromInv[indexofInv].y;
		}

	}
}

__global__ void scale(int samples, hipfftComplex *data, int fftsize)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < samples; i += stride) {
		data[i].x = data[i].x / float(fftsize);
		data[i].y = data[i].y / float(fftsize);

	}
}

__global__ void maskAndShift(char *devicedata, hipComplex *Dcomplexdata, int totalBytes)
{
	unsigned char k, aux;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < totalBytes; i += stride) {
		k = (unsigned char)(devicedata[i]);

		aux = k & ((unsigned)1);
		aux = aux >> 0;
		Dcomplexdata[i * 4 + 0].x = float(2 * (aux)-1);

		aux = k & ((unsigned)(1 << 1));
		aux = aux >> 1;

		Dcomplexdata[i * 4 + 0].y = float(2 * (aux)-1);
		aux = k & ((unsigned)(1 << 2));
		aux = aux >> 2;
		Dcomplexdata[i * 4 + 1].x = float(2 * (aux)-1);
		aux = k & ((unsigned)(1 << 3));
		aux = aux >> 3;
		Dcomplexdata[i * 4 + 1].y = float(2 * (aux)-1);
		aux = k & ((unsigned)(1 << 4));
		aux = aux >> 4;
		Dcomplexdata[i * 4 + 2].x = float(2 * (aux)-1);
		aux = k & ((unsigned)(1 << 5));
		aux = aux >> 5;
		Dcomplexdata[i * 4 + 2].y = float(2 * (aux)-1);
		aux = k & ((unsigned)(1 << 6));
		aux = aux >> 6;
		Dcomplexdata[i * 4 + 3].x = float(2 * (aux)-1);
		aux = k & ((unsigned)(1 << 7));
		aux = aux >> 7;
		Dcomplexdata[i * 4 + 3].y = float(2 * (aux)-1);

	}
}


__global__ void copyInt2Float(__int16 *deviceIntData, hipComplex *deviceFloatData, int length) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < length/2; i += stride) {
		deviceFloatData[i].x = float(deviceIntData[2*i]);
		deviceFloatData[i].y = float(deviceIntData[2*i+1]);
	}


}
