//==========================================================================
// Author: Oriol Cervelló (oriol.cn [at] protonmail.com) 
//==========================================================================
// License: GNU GPLv3.0
// Copyright (C) 2019  Oriol Cervelló
//
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
// 
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
// 
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.
//==========================================================================
#include"Hostfunc.cuh"
#include "TextParser.cuh"


//INPUT CONFIG PARSER FUNCTIONS

void readConfig(const char *configFileName, int numofDataLines, int *fftsize, int *numofFFts, int *overlap, int *fSampling, int *blockSize, int *peakRangeStd, int *peakSamplesToSave,
	int* quantOfAverIncoh, int *dataOffsetBeg, int *dataOffsetEnd, float *doppler, string *fileNames,string *fileRefNames, int *ddmRes, int *ddmQuant,bool *interfer
,int *dataOffsetBegInterferometric,int *samplesAvoidMaxs,string *resultDirectory,bool *writeoutputs, int *typeOfDataline,int * dataOffsetEndInterferometric) {

	TextParser t(configFileName);
	TextParserSafeCall(t.seek("*WRITEWAVEFORM"));
	*writeoutputs = t.getint();
	TextParserSafeCall(t.seek("*FFTSIZE"));
	*fftsize = t.getint();
	TextParserSafeCall(t.seek("*NUMOFFFTS"));
	*numofFFts = t.getint();
	TextParserSafeCall(t.seek("*QUANTINCOHAVER"));
	*quantOfAverIncoh = t.getint();
	TextParserSafeCall(t.seek("*OVERLAP"));
	*overlap = t.getint();
	TextParserSafeCall(t.seek("*FSAMPLING"));
	*fSampling = t.getint();
	TextParserSafeCall(t.seek("*BLOCKSIZE"));
	*blockSize = t.getint();
	TextParserSafeCall(t.seek("*INTERFEROMETIC"));
	*interfer = t.getint();
	TextParserSafeCall(t.seek("*SAMPLESAVOIDMAX"));
	*samplesAvoidMaxs = t.getint();
	TextParserSafeCall(t.seek("*PEAKRANGESTD"));
	*peakRangeStd = t.getint();
	TextParserSafeCall(t.seek("*PEAKSAMPLESTOSAVE"));
	*peakSamplesToSave = t.getint();
	if (*interfer == false) {
		TextParserSafeCall(t.seek("*REFFILENAME"));
		fileRefNames[0] = t.getword(); 
	}
	TextParserSafeCall(t.seek("*RESULTSDIRECTORY"));
	*resultDirectory = t.getword();
	TextParserSafeCall(t.seek("*DDMFREQRES"));
	*ddmRes = t.getint();
	TextParserSafeCall(t.seek("*DDMNUMQUANT"));
	*ddmQuant = t.getint();

	if (*ddmQuant % 2 != 1) {
		cout << "ERROR: DDM QUANT has to be odd: 1(original)+2n(symethric)\n";
		exit(1);

	}


	TextParserSafeCall(t.seek("*QUANTDATALINES"));
	if (t.getint() != numofDataLines) {
		cout << "diferent num of Data lines in the file than declared on arguments \n  To execute enter arguments: NameconfigFile.ASE NumofDataLines\n";
		exit(1);
	}

	for (int i = 0; i < numofDataLines; i++) {
		TextParserSafeCall(t.seek("*DATALINE"));

		typeOfDataline[i] = t.getint();
		fileNames[i] = t.getword();
		dataOffsetBeg[i] = t.getint();
		dataOffsetEnd[i] = t.getint();
		doppler[i] = t.getfloat();
		if (*interfer == true) {
			fileRefNames[i] = t.getword();
			dataOffsetBegInterferometric[i]= t.getint();
			dataOffsetEndInterferometric[i]= t.getint();
		}
	}
}

void checkInputConfig(int argc, const char **argv, int numofDataLines, int fftsize, int numofFFts, int overlap, int fSampling,  int blockSize, int peakRangeStd, int peakSamplesToSave,
	int quantOfAverIncoh,  int *dataOffsetBeg, int *dataOffsetEnd, float *doppler, string *fileNames, string *fileRefNames, int ddmRes, int ddmQuant, bool interfer
, int *dataOffsetBegInterferometric, int samplesAvoidMaxs,string resultDirectory, bool writeoutputs, int *typeOfDataline, int * dataOffsetEndInterferometric) {

	if (argc != 3) {
		cout << "Error: Wrong number of arguments\n"; 
		exit(0);
	}

	cout << "\n" << "-ARGS: " << argc << "\n";
	cout << "First: " << argv[0] << "\n";
	cout << "Second: " << argv[1] << "\n";
	cout << "Third: " << argv[2] << "\n\n";

	cout << "-INPUTS:\n";
	cout << "Write outputs: " << writeoutputs << "\n";
	cout << "FFT Size: " << fftsize << "\n";
	cout << "Num. of FFT: " << numofFFts << "\n";
	cout << "Overlap: " << overlap << "\n";
	cout << "FSampling: " << fSampling << "\n";
	cout << "Quant of averg Inch.: " << quantOfAverIncoh << "\n";
	cout << "Blok Size: " << blockSize << "\n";
	cout << "Interferometric: " << interfer << "\n";
	cout << "Samples avoid MAxs: " << samplesAvoidMaxs << "\n";
	cout << "Peak samples for the std: " << peakRangeStd << "\n";
	cout << "Peak samples to save: " << peakSamplesToSave << "\n";
	if (interfer == false) {
		cout << "Ref File Name: " << fileRefNames[0] << "\n";
	}
	cout << "result directory: " << resultDirectory << "\n";
	cout << "DDM Res: " << ddmRes << "\n";
	cout << "DDM Quant: " << ddmQuant << "\n";


	cout << "Num of data lines: " << numofDataLines << "\n";
	cout << "Data lines: \n";
	for (int i = 0; i < numofDataLines; i++) {
		cout << typeOfDataline[i] << "  ";
		cout << fileNames[i] << "  ";
		cout << dataOffsetBeg[i] << "  ";
		cout << dataOffsetEnd[i] << "  ";
		cout << doppler[i] << "  ";
		if (interfer == true) {
			cout << fileRefNames[i] << "  ";
			cout << dataOffsetBegInterferometric[i] << " ";
			cout << dataOffsetEndInterferometric[i] << "\n";
		}
		else {
			cout << "\n";
		}
	}

}

//PREPARE DATA FUNCTIONS

void prepareReference(int fftsize, int overlap,int blockSize ,hipfftComplex *hostDataFile2, hipfftComplex *deviceDataFile2,string fileRefName) {
	
	readdata(fftsize - overlap, 0, hostDataFile2, fileRefName);
	hipfftHandle planref;
	CudaSafeCall(hipMemcpy(deviceDataFile2, hostDataFile2, sizeof(hipfftComplex)*(fftsize - overlap), hipMemcpyHostToDevice));
	hipDeviceSynchronize();
	if (overlap > 0) {
		int numBlocks = (fftsize + blockSize - 1) / blockSize;
		extendRefSignal << <numBlocks, blockSize >> > (fftsize, deviceDataFile2, fftsize - overlap);
		CudaCheckError();
	}
	planfftFunction(fftsize, 1, 0, &planref);
	hipDeviceSynchronize();
	cufftSafeCall(hipfftExecC2C(planref, deviceDataFile2, deviceDataFile2, HIPFFT_FORWARD));
	hipDeviceSynchronize();
	cufftSafeCall(hipfftDestroy(planref));

}


void prepareData( int *dataOffsetEnd,int *dataOffsetBeg, int bytesToRead, char *hostBytesOfData, string *fileDataNames,
	char *deviceBytesOfData, int blockSize, int ddmQuant, int samplesOfSignal, int samplesWithOverlap, hipfftComplex *deviceDataFile1
     ,int numofFFTs, int fftsize, hipfftComplex *hostDataFile1, chrono::nanoseconds *elapsed_read, chrono::nanoseconds *mask_elapsed
	,chrono::nanoseconds *extenddop_elapsed, int * typeOfDataline,int ind) {
	
	auto begin = std::chrono::high_resolution_clock::now();
	//READ DATA
	//readdata(dataOffsetEnd[i]-dataOffsetBeg[i], dataOffsetBeg[i], hostDataFile1, fileDataNames[i]);
	if (typeOfDataline[ind] == 1) {
		readRealData(dataOffsetEnd[ind] - dataOffsetBeg[ind], dataOffsetBeg[ind], bytesToRead, hostBytesOfData, fileDataNames[ind]);
	}
	else {
		readRealData2files(dataOffsetEnd[ind] - dataOffsetBeg[ind], dataOffsetEnd[ind + 1] - dataOffsetBeg[ind + 1], dataOffsetBeg[ind], dataOffsetBeg[ind + 1], bytesToRead, hostBytesOfData, fileDataNames[ind], fileDataNames[ind + 1]);
	}
	//CudaSafeCall(hipMemcpy(deviceDataFile1, hostDataFile1, sizeof(hipfftComplex)*samplesOfSignal, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(deviceBytesOfData, hostBytesOfData, sizeof(char)*bytesToRead, hipMemcpyHostToDevice));
	hipDeviceSynchronize();
	*elapsed_read = chrono::high_resolution_clock::now() - (begin);

	//MASK AND SHIFT
	auto maskbeg = std::chrono::high_resolution_clock::now();
	int numBlocks = (bytesToRead + blockSize - 1) / blockSize;
	maskAndShift << <numBlocks, blockSize >> > (deviceBytesOfData, deviceDataFile1, bytesToRead);
	CudaCheckError();
	hipDeviceSynchronize();
	*mask_elapsed = chrono::high_resolution_clock::now() - maskbeg;

	//EXTEND FOR DOPPLER
	auto extenddopbeg = std::chrono::high_resolution_clock::now();
	if (ddmQuant > 1) {
		numBlocks = (samplesOfSignal + blockSize - 1) / blockSize;
		extendRefSignal << <numBlocks, blockSize >> > (samplesWithOverlap, deviceDataFile1, numofFFTs * fftsize);
		CudaCheckError();
		hipDeviceSynchronize();
	}
	*extenddop_elapsed = chrono::high_resolution_clock::now() - extenddopbeg;


}

//FFT PLANS FUNCTIONS

void planfftFunction(int fftsize, int numofFFTs, int overlap, hipfftHandle *plan) {

	int rank = 1;                           // --- 1D FFTs
	int n[] = { fftsize };                 // --- Size of the Fourier transform
	int istride = 1, ostride = 1;           // --- Distance between two successive input/output elements
	int idist = fftsize - overlap, odist = fftsize;// (DATASIZE / 2 + 1); // --- Distance between batches
	int inembed[] = { 0 };                  // --- Input size with pitch (ignored for 1D transforms)
	int onembed[] = { 0 };                  // --- Output size with pitch (ignored for 1D transforms)
	int batch = numofFFTs;// numofFFTs;                      // --- Number of batched executions
	cufftSafeCall(hipfftPlanMany(plan, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch));

}

void planifftFunction(int fftsize, int numofFFTs, int overlap, hipfftHandle *plan) {
	
	int rank = 1;                           // --- 1D FFTs
	int n[] = { fftsize };                 // --- Size of the Fourier transform
	int istride = 1, ostride = 1;           // --- Distance between two successive input/output elements
	int idist = fftsize, odist = fftsize - overlap;// (DATASIZE / 2 + 1); // --- Distance between batches
	int inembed[] = { 0 };                  // --- Input size with pitch (ignored for 1D transforms)
	int onembed[] = { 0 };                  // --- Output size with pitch (ignored for 1D transforms)
	int batch = numofFFTs;// numofFFTs;                      // --- Number of batched executions
	cufftSafeCall(hipfftPlanMany(plan, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch));

}

size_t planMemEstimate(int fftsize, int numofFFTs, int overlap) {

	int rank = 1;                           // --- 1D FFTs
	int n[] = { fftsize };                 // --- Size of the Fourier transform
	int istride = 1, ostride = 1;           // --- Distance between two successive input/output elements
	int idist = fftsize, odist = fftsize - overlap;// (DATASIZE / 2 + 1); // --- Distance between batches
	int inembed[] = { 0 };                  // --- Input size with pitch (ignored for 1D transforms)
	int onembed[] = { 0 };                  // --- Output size with pitch (ignored for 1D transforms)
	int batch = numofFFTs;// numofFFTs;                      // --- Number of batched executions
	size_t workSize;
	cufftSafeCall(hipfftEstimateMany( rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch,&workSize));

	cout << "cufft plan aprox buffer: " << workSize<< " bytes\n";
	return workSize;
}

//STATISTICS FUNCTIONS

void maxCompute(int numofIncoherentSums, Npp32f *deviceDataIncoherentSum, int fftsize, Npp32f *deviceArrayMaxs,
	 int *deviceArrayPos, Npp8u * pDeviceBuffer, int samplesAvoidMax) {

	for (int i = 0; i < numofIncoherentSums; i++) {

		nppsMaxIndx_32f(&deviceDataIncoherentSum[i*fftsize+(samplesAvoidMax)], fftsize+ samplesAvoidMax, &deviceArrayMaxs[i], &deviceArrayPos[i], pDeviceBuffer);
	}
}

void stdCompute(int numofIncoherentSums, Npp32f *dataIncoherentSum, int fftsize,
	Npp32f *deviceArraystd, int *arrayPos, Npp8u * pStdDeviceBuffer, int peakRange,int stdLength, Npp32f *devicearrayMean) {

	int leftPeakIndex, rightPeakIndex;
	//stdLength = (fftsize / 2) - ((peakRange) / 2)-1;
	for (int i = 0; i < numofIncoherentSums; i++) {
		
		leftPeakIndex = arrayPos[i] - peakRange/2;
		rightPeakIndex = arrayPos[i] + peakRange/2;
		
		if (rightPeakIndex >= fftsize) {//case 2
			rightPeakIndex = rightPeakIndex % fftsize;
			//stdLength = leftPeakIndex - rightPeakIndex;
			nppsMeanStdDev_32f(&dataIncoherentSum[i*fftsize+ rightPeakIndex], stdLength,&devicearrayMean[i],&deviceArraystd[i], pStdDeviceBuffer);
		}
		else if (leftPeakIndex < 0) {//case 3
			leftPeakIndex = fftsize + leftPeakIndex;
			//stdLength = leftPeakIndex-rightPeakIndex ;
			nppsMeanStdDev_32f(&dataIncoherentSum[i*fftsize + rightPeakIndex], stdLength, &devicearrayMean[i], &deviceArraystd[i], pStdDeviceBuffer);
		}
		else {//case 1
			if (arrayPos[i] < fftsize / 2) {
				//stdLength = fftsize- rightPeakIndex;
				nppsMeanStdDev_32f(&dataIncoherentSum[i*fftsize + rightPeakIndex], stdLength, &devicearrayMean[i], &deviceArraystd[i], pStdDeviceBuffer);
			}
			else {
				//stdLength = leftPeakIndex;
				nppsMeanStdDev_32f(&dataIncoherentSum[i*fftsize], stdLength, &devicearrayMean[i], &deviceArraystd[i], pStdDeviceBuffer);
			}			
		}		
	}
}

