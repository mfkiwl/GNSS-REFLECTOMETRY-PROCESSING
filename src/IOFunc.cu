//==========================================================================
// Author: Oriol Cervelló (oriol.cn [at] protonmail.com) 
//==========================================================================
// License: GNU GPLv3.0
// Copyright (C) 2019  Oriol Cervelló
//
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
// 
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
// 
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.
//==========================================================================
#include "IOFunc.cuh"
//READ FUNCTIONS

void readdata(int length, int offsetFromBeg, hipfftComplex *data, string name)
{
	ifstream myfile;
	myfile.open(name, ios::binary);
	//float num1,num2;

	if (myfile.is_open())
	{
		myfile.seekg(offsetFromBeg * sizeof(hipfftComplex));

		myfile.read((char*)data, length * sizeof(hipfftComplex));
		/*int k = 0;
		while (k < length)
		{

			myfile.read((char*)&num1, sizeof(num1));
			myfile.read((char*)&num2, sizeof(num2));
			data[k].x = num1;
			data[k].y =  num2;
			k++;
		}*/
		myfile.close();
	}
	else { 
		cout << "ERROR: Unable to open file of floats for reading " << name << "\n";
		exit(1);
	}
}

void readRealData(int length, int offsetFromBeg, int bytesToRead, char *data, string name)
{
	if (length > bytesToRead) {
		cout << "Error: iput length bigger than bytesToRead\n";
		exit(0);
	}

	ifstream myfile;
	myfile.open(name, ios::binary);
	if (myfile.is_open())
	{
		myfile.seekg(offsetFromBeg * sizeof(char));
		myfile.read(data, length);

		myfile.close();
		if (length < bytesToRead) {
			cout << "Warning: length smaller than bytesToRead, " << bytesToRead - length << " Bytes filled with 0 \n Last/s incoherents will be incomplete \n";
			memset(&data[length], 0, bytesToRead - length);
		}
	}
	else { 
		cout << "ERROR: Unable to open file of Real Data for reading " << name << "\n";
		exit(1);
	}
}


void readRealData2files(int length1, int length2, int offsetFromBeg1, int offsetFromBeg2, int bytesToRead, char *data, string name1, string name2)
{
	if (length1 > bytesToRead || length2 > bytesToRead) {
		cout << "Error: iput length bigger than bytesToRead\n";
		exit(0);
	}

	ifstream myfile;
	myfile.open(name1, ios::binary);
	if (myfile.is_open())
	{
		myfile.seekg(offsetFromBeg1 * sizeof(char));
		myfile.read(data, length1);

		myfile.close();
		
	}
	else {
		cout << "ERROR: Unable to open file of Real Data for reading " << name1 << "\n";
		exit(1);
	}

	myfile.open(name2, ios::binary);
	if (myfile.is_open())
	{
		myfile.seekg(offsetFromBeg2 * sizeof(char));
		myfile.read(&data[length1], length2);

		myfile.close();

	}
	else {
		cout << "ERROR: Unable to open file of Real Data for reading " << name2 << "\n";
		exit(1);
	}


	if (length1+length2 < bytesToRead) {
		cout << "Warning: length smaller than bytesToRead, " << bytesToRead - length1-length2 << " Bytes filled with 0 \n Last/s incoherents will be incomplete \n";
		memset(&data[length1+length2], 0, bytesToRead - (length1 + length2));
	}
}




void readdataInt(int length, int offsetFromBeg, __int16 *data, string name)
{
	
	ifstream myfile;
	myfile.open(name, ios::binary);
	//float num1,num2;

	if (myfile.is_open())
	{
		myfile.seekg(offsetFromBeg * sizeof(__int16));

		myfile.read((char*)data, 2* length * sizeof(__int16));
		
		myfile.close();
	}
	else {
		cout << "ERROR: Unable to open file of Ints16 for reading " << name << "\n"; 
		exit(1);
	}

}

//WRITE FUNCTIONS

void writeIncoh(int N, hipComplex *data1, string name) {

	ofstream myfile;
	myfile.open(name, ios::binary);
	if (myfile.is_open())
	{
		for (int ii = 0; ii < N / 2; ii++)
		{
			myfile.write((char*)&data1[ii].x, sizeof(float));
			myfile.write((char*)&data1[ii].y, sizeof(float));
		}
		myfile.close();
	}

	else { 
		cout << "ERROR: Unable to open file of incoh for writting " << name << "\n"; 
		exit(1);
	}
}

void writeMaxs(int N, Npp32f *dataMaxValue, int *dataMaxPos, Npp32f *hostarrayStd, Npp32f *hostarrayMean, float doppler, string name, int iteration, int ddmRes,
	int ddmQuant, int origIncohNum) {
	float freq, aux;
	ofstream myfile;
	myfile.open(name, ios::binary);//
	if (myfile.is_open())
	{
		for (int ii = 0; ii < N; ii++)
		{
			aux = float(dataMaxPos[ii]);
			freq = doppler - (ddmRes * (ddmQuant / 2)) + ((ii / (origIncohNum))*(ddmRes));
			myfile.write((char*)&aux, sizeof(float));
			myfile.write((char*)&dataMaxValue[ii], sizeof(float));
			myfile.write((char*)&hostarrayMean[ii], sizeof(float));
			myfile.write((char*)&hostarrayStd[ii], sizeof(float));
			myfile.write((char*)&freq, sizeof(float));
			//myfile <<float( dataMaxPos[ii])<< " "<<dataMaxValue[ii] << " " << hostarrayMean[ii] << " " << hostarrayStd[ii] << " " << freq << " ";
		}
		myfile.close();
	}

	else {
		cout << "ERROR: Unable to open file of Maxs " << name << "\n";
		exit(1);
	}
}

void writedata(int N, hipfftComplex *data1, string name) {

	ofstream myfile;
	myfile.open(name, ios::binary);
	if (myfile.is_open())
	{
		myfile.write((char*)data1, N * sizeof(hipfftComplex));
		/*for (int ii = 0; ii < N; ii++)
		{

			myfile.write((char*)&data1[ii].x, sizeof(float));
			myfile.write((char*)&data1[ii].y, sizeof(float));
		}*/
		myfile.close();
	}

	else {
		cout << "ERROR: Unable to open file of data to write " << name << "\n";
		exit(1);
	}
}

void writetime(int N, string name, long long *readtime, long long *writetime, long long *looptime
	, long long *mask_elapsed_secs, long long *extenddop_elapsed_secs, long long *doppler_elapsed_secs,
	long long *fft_elapsed_secs, long long *mult_elapsed_secs, long long *ifft_elapsed_secs,
	long long *incho_elapsed_secs, long long *max_elapsed_secs, long long *savep_elapsed_secs, long long *std_elapsed_secs) {

	ofstream myfile;
	myfile.open(name);
	if (myfile.is_open())
	{
		myfile << "Atempt\t\tReadT.\t\tMask\t\tExtend\t\tDoppler\t\tFFT\t\tMul\t\tIFFT\t\tIncoh\t\tMax\t\tSaveP.\t\tSTD\t\tWriteT.\t\tLoopT." << "\n";
		for (int ii = 0; ii < N; ii++)
		{
			myfile << ii << "\t\t" << readtime[ii] << "\t\t" << mask_elapsed_secs[ii] << "\t\t"
				<< extenddop_elapsed_secs[ii] << "\t\t" << doppler_elapsed_secs[ii] << "\t\t"
				<< fft_elapsed_secs[ii] << "\t\t" << mult_elapsed_secs[ii] << "\t\t"
				<< ifft_elapsed_secs[ii] << "\t\t"
				<< incho_elapsed_secs[ii] << "\t\t" << max_elapsed_secs[ii] << "\t\t"
				<< savep_elapsed_secs[ii] << "\t\t" << std_elapsed_secs[ii] << "\t\t"
				<< writetime[ii] << "\t\t" << looptime[ii] << "\n";

		}
		myfile.close();
	}

	else {
		cout << "ERROR: Unable to open file of times " << name << "\n";
		exit(1);
	}
}
