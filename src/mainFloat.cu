//==========================================================================
// Author: Oriol Cervelló (oriol.cn [at] protonmail.com) 
//==========================================================================
// License: GNU GPLv3.0
// Copyright (C) 2019  Oriol Cervelló
//
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
// 
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
// 
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.
//==========================================================================
#ifndef LIBRARIES
#define LIBRARIES
#include <math.h>
#include <fstream>
#include <iostream>
#include <string>
#include <stdio.h>
#include <hipfft/hipfft.h>
#include <ctime>
#include <chrono>
#include <npps.h>
#define PI 3.14159265
using namespace std;
#endif

#include"HostFunc.cuh"
#include "GlobalFunc.cuh"
#include "IOFunc.cuh"





void prepareDataFloat( int dataOffsetEnd,int dataOffsetBeg, int bytesToRead, char *hostBytesOfData, string fileDataNames,
	char *deviceBytesOfData, int blockSize, int ddmQuant, int samplesOfSignal, int samplesWithOverlap, hipfftComplex *deviceDataFile1
     ,int numofFFTs, int fftsize, hipfftComplex *hostDataFile1, chrono::nanoseconds *elapsed_read, chrono::nanoseconds *mask_elapsed
	,chrono::nanoseconds *extenddop_elapsed) {
	
	auto begin = std::chrono::high_resolution_clock::now();
	//READ DATA
	readdata(dataOffsetEnd-dataOffsetBeg, dataOffsetBeg, hostDataFile1, fileDataNames);
	//readRealData(dataOffsetEnd - dataOffsetBeg, dataOffsetBeg, bytesToRead, hostBytesOfData, fileDataNames);

	CudaSafeCall(hipMemcpy(deviceDataFile1, hostDataFile1, sizeof(hipfftComplex)*samplesOfSignal, hipMemcpyHostToDevice));
	//CudaSafeCall(hipMemcpy(deviceBytesOfData, hostBytesOfData, sizeof(char)*bytesToRead, hipMemcpyHostToDevice));
	hipDeviceSynchronize();
	*elapsed_read = chrono::high_resolution_clock::now() - (begin);

	//MASK AND SHIFT
	auto maskbeg = std::chrono::high_resolution_clock::now();
	//int numBlocks = (bytesToRead + blockSize - 1) / blockSize;
	//maskAndShift << <numBlocks, blockSize >> > (deviceBytesOfData, deviceDataFile1, bytesToRead);
	//CudaCheckError();
	//hipDeviceSynchronize();
	*mask_elapsed = chrono::high_resolution_clock::now() - maskbeg;

	//EXTEND FOR DOPPLER
	auto extenddopbeg = std::chrono::high_resolution_clock::now();
	if (ddmQuant > 1) {
		int numBlocks = (samplesOfSignal + blockSize - 1) / blockSize;
		extendRefSignal << <numBlocks, blockSize >> > (samplesWithOverlap, deviceDataFile1, numofFFTs * fftsize);
		CudaCheckError();
		hipDeviceSynchronize();
	}
	*extenddop_elapsed = chrono::high_resolution_clock::now() - extenddopbeg;


}



int main(int argc, const char* argv[]) {
	hipDeviceReset();//reset device

	//READ CONFIG FILE
	int fftsize, fSampling, numofFFTs, overlap, quantofAverageIncoherent, blockSize, peakRangeStd, peakSamplesToSave,
		samplesAvoidMax,ddmRes, ddmQuant;
	int const numofDataLines = atoi(argv[2]);//substitut d'iterations
	bool interferometic,writeoutputs=1;
	string *fileDataNames, *fileRefName, resultDirectory;
	int *dataOffsetBeg, *dataOffsetEnd,*dataOffsetBegInterferometric;
	float *doppler;

	fileRefName = new string[numofDataLines];
	fileDataNames = new string[numofDataLines];
	dataOffsetBeg = new int[numofDataLines];
	dataOffsetBegInterferometric = new int[numofDataLines];
	dataOffsetEnd = new int[numofDataLines];
	doppler = new float[numofDataLines];

	readConfig(argv[1], numofDataLines, &fftsize, &numofFFTs, &overlap, &fSampling, &blockSize, &peakRangeStd, &peakSamplesToSave, &quantofAverageIncoherent, dataOffsetBeg, dataOffsetEnd, doppler, fileDataNames, fileRefName, &ddmRes, &ddmQuant,&interferometic,dataOffsetBegInterferometric, &samplesAvoidMax, &resultDirectory,&writeoutputs);
	checkInputConfig(argc, argv, numofDataLines, fftsize, numofFFTs, overlap, fSampling, blockSize, peakRangeStd, peakSamplesToSave, quantofAverageIncoherent, dataOffsetBeg, dataOffsetEnd, doppler, fileDataNames, fileRefName, ddmRes, ddmQuant, interferometic, dataOffsetBegInterferometric, samplesAvoidMax, resultDirectory);

	//OTHER DECLARATIONS
	int  originalSamplesOfSignal = (numofFFTs * (fftsize - overlap)) + overlap;//samples of complex data
	int samplesOfSignal = originalSamplesOfSignal *ddmQuant;//samples of complex data
	int bytesToRead = originalSamplesOfSignal /4;
	if (originalSamplesOfSignal % 4 != 0) { cout << "Warning bytesToRead rounded toward negative infinity: samplesOfSignal%4!=0 \n "; }
	int samplesWithOverlap= (numofFFTs * fftsize)*ddmQuant;//total samples needed
	if(samplesOfSignal > samplesWithOverlap){ samplesWithOverlap = samplesOfSignal;}
	int inchoerentNumofFFT = (numofFFTs/ quantofAverageIncoherent)*ddmQuant;
	if (numofFFTs % quantofAverageIncoherent != 0) {
		cout << "Error: numofFFTs / quantofAverageIncoherent != 0\n ";
		exit(-1);
	}

	string outputName;
	int numBlocks, nMaxBufferSize,nStdBufferSize,i,k,samplesDoppler= samplesOfSignal;
	int stdLength = (fftsize / 2) - ((peakSamplesToSave) / 2) - 1;
	unsigned long long samplePhaseMantain;
	//int checkMax
	if (ddmQuant > 1) {
		samplesDoppler = samplesWithOverlap;
	}


	char *hostBytesOfData, *deviceBytesOfData;
	int *devicearrayPos,*hostarrayPos;
	hipfftComplex *deviceDataFile1, *deviceDataFile2, *hostDataFile1, *hostDataFile2, *deviceDataToSave;
	Npp32f *deviceIncoherentSum, *devicearrayMaxs, *devicearrayStd,*devicearrayMean,*hostarrayMean, *hostarrayMaxs, *hostarrayStd;
	Npp8u *pStdDeviceBuffer,*pMaxDeviceBuffer;

	hipfftHandle plan;
	
	long long *read_elapsed_secs,*write_elapsed_secs, *elapsed_secs, *mask_elapsed_secs, *doppler_elapsed_secs, 
		 *fft_elapsed_secs, *mult_elapsed_secs,*ifft_elapsed_secs, *extenddop_elapsed_secs, *incho_elapsed_secs
		, *max_elapsed_secs, *savep_elapsed_secs, *std_elapsed_secs;
	chrono::nanoseconds elapsed_read, mask_elapsed, extenddop_elapsed;

	int device2quant;
	if (interferometic == true) {
		device2quant = samplesWithOverlap;
	}
	else
	{
		device2quant = fftsize;
	}


	//ALLOCATE
	read_elapsed_secs = new long long[numofDataLines];
	mask_elapsed_secs = new long long[numofDataLines];
	doppler_elapsed_secs = new long long[numofDataLines];
	fft_elapsed_secs = new long long[numofDataLines];
	mult_elapsed_secs = new long long[numofDataLines];
	ifft_elapsed_secs = new long long[numofDataLines];
	extenddop_elapsed_secs = new long long[numofDataLines];
	incho_elapsed_secs = new long long[numofDataLines];
	max_elapsed_secs = new long long[numofDataLines];
	savep_elapsed_secs = new long long[numofDataLines];
	std_elapsed_secs = new long long[numofDataLines];
	write_elapsed_secs = new long long[numofDataLines];
	elapsed_secs = new long long[numofDataLines];

	hostBytesOfData = (char *)malloc(sizeof(char) * 1);
	hostarrayPos = new int[inchoerentNumofFFT];
	hostarrayMaxs = new Npp32f[inchoerentNumofFFT];
	hostarrayStd = new Npp32f[inchoerentNumofFFT];
	hostarrayMean = new Npp32f[inchoerentNumofFFT];
	hostDataFile1 = (hipfftComplex *)malloc(sizeof(hipfftComplex) * samplesWithOverlap);
	hostDataFile2 = (hipfftComplex *)malloc(sizeof(hipfftComplex) * device2quant);
	CudaSafeCall(hipMalloc(&deviceBytesOfData, sizeof(char)*1));
	CudaSafeCall(hipMalloc(&deviceDataFile1, sizeof(hipfftComplex)*samplesWithOverlap));
	CudaSafeCall(hipMalloc(&deviceDataToSave, sizeof(hipfftComplex)*peakSamplesToSave*numofFFTs*ddmQuant));
	CudaSafeCall(hipMalloc(&deviceDataFile2, sizeof(hipfftComplex)*device2quant));
	CudaSafeCall(hipMalloc(&deviceIncoherentSum, sizeof(Npp32f)*inchoerentNumofFFT*fftsize));
	CudaSafeCall(hipMalloc(&devicearrayPos, sizeof(int)*inchoerentNumofFFT));
	CudaSafeCall(hipMalloc(&devicearrayMaxs, sizeof(Npp32f)*inchoerentNumofFFT));
	CudaSafeCall(hipMalloc(&devicearrayStd, sizeof(Npp32f)*inchoerentNumofFFT));
	CudaSafeCall(hipMalloc(&devicearrayMean, sizeof(Npp32f)*inchoerentNumofFFT));
	nppsMeanStdDevGetBufferSize_32f(stdLength, &nStdBufferSize);
	CudaSafeCall(hipMalloc((void **)(&pStdDeviceBuffer), nStdBufferSize));
	nppsMaxGetBufferSize_32f(fftsize, &nMaxBufferSize);
	CudaSafeCall(hipMalloc((void **)(&pMaxDeviceBuffer), nMaxBufferSize));
	hipDeviceSynchronize();
	
	//MMEMORY INFO
	size_t freeMem, totalMem;
	hipMemGetInfo(&freeMem, &totalMem);
	cout<< "\n-MEMORY: \n";
	cout<< "Total GPU mem: "<< totalMem <<" bytes\n";
	size_t planBuffer = planMemEstimate(fftsize, numofFFTs, overlap);
	long long allocatedMem =  sizeof(hipfftComplex)*samplesWithOverlap + nMaxBufferSize +
		sizeof(hipfftComplex)*peakSamplesToSave*numofFFTs + sizeof(hipfftComplex)*device2quant + sizeof(Npp32f)*inchoerentNumofFFT*fftsize+
		sizeof(Npp32f)*inchoerentNumofFFT*fftsize + sizeof(int)*inchoerentNumofFFT + sizeof(Npp32f)*inchoerentNumofFFT + sizeof(Npp32f)*inchoerentNumofFFT + nStdBufferSize;
	cout << "GPU mem allocated: " << allocatedMem <<" bytes\n";
	cout << "GPU total aprox mem used: " << allocatedMem+ planBuffer <<" bytes\n\n";
	
	//READ, EXTEND AND FFT OF REF SIGNAL
	if (interferometic == false) {
		prepareReference(fftsize, overlap, blockSize, hostDataFile2, deviceDataFile2, fileRefName[0]);
		delete[] fileRefName;
		delete[] dataOffsetBegInterferometric;
	}

	//LOOP
	for (i = 0; i < numofDataLines; i++) {
		
		auto begin = std::chrono::high_resolution_clock::now();
		//READ, MASK, AND EXTEND
		prepareDataFloat(dataOffsetEnd[i], dataOffsetBeg[i], bytesToRead, hostBytesOfData, fileDataNames[i],
			deviceBytesOfData, blockSize, ddmQuant, samplesOfSignal, samplesWithOverlap, deviceDataFile1
			, numofFFTs, fftsize, hostDataFile1,&elapsed_read, &mask_elapsed
			, &extenddop_elapsed);

		if (interferometic == true) {
			chrono::nanoseconds elapsed_read_inter, mask_elapsed_inter, extenddop_elapsed_inter;
			prepareDataFloat( dataOffsetEnd[i]+(dataOffsetBegInterferometric[i]-dataOffsetBeg[i]), dataOffsetBegInterferometric[i],
				bytesToRead, hostBytesOfData, fileRefName[i],deviceBytesOfData, blockSize, ddmQuant, samplesOfSignal,
				samplesWithOverlap, deviceDataFile2, numofFFTs, fftsize, hostDataFile2,&elapsed_read_inter, &mask_elapsed_inter
				, &extenddop_elapsed_inter);
			elapsed_read += elapsed_read_inter;
			mask_elapsed += mask_elapsed_inter;
			extenddop_elapsed += extenddop_elapsed_inter;
		}

		//CHECK: RAW DATA 
		//CudaSafeCall(hipMemcpy(hostDataFile1, deviceDataFile1, sizeof(hipfftComplex)*(dataOffsetEnd[i] - dataOffsetBeg[i])*4, hipMemcpyDeviceToHost));
		//hipDeviceSynchronize();
		//writedata((dataOffsetEnd[i] - dataOffsetBeg[i])*4, hostDataFile1, "results/rawdata.bin");

		//MULTIPLY BY DOPPLER
		auto dopplerbeg = std::chrono::high_resolution_clock::now();
		
		samplePhaseMantain = (unsigned long long(i) * unsigned long long(fftsize*numofFFTs));
		numBlocks = (samplesDoppler + blockSize - 1) / blockSize;
		applyDoppler << <numBlocks, blockSize >> > (samplesDoppler, deviceDataFile1, doppler[i], fSampling, samplePhaseMantain, fftsize * numofFFTs, ddmQuant, ddmRes, fftsize);
		CudaCheckError();
		hipDeviceSynchronize();
	
		auto doppler_elapsed = chrono::high_resolution_clock::now() - dopplerbeg;
		
		//CHECK: doppler (only for printing doppler)
		//CudaSafeCall(hipMemcpy(hostDataFile1, deviceDataFile1, sizeof(hipfftComplex)*samplesWithOverlap, hipMemcpyDeviceToHost));
		//hipDeviceSynchronize();
		//writedata(samplesWithOverlap, hostDataFile1, "results/dopplerout.bin");

		//FFT
		auto fftbeg = std::chrono::high_resolution_clock::now();
		planfftFunction(fftsize, numofFFTs, overlap, &plan);
		hipDeviceSynchronize();
		for (k = 0; k < ddmQuant; k++) {
			cufftSafeCall(hipfftExecC2C(plan, &deviceDataFile1[k*(numofFFTs * fftsize)], &deviceDataFile1[k*(numofFFTs * fftsize)], HIPFFT_FORWARD));
			hipDeviceSynchronize();
			if (interferometic == true) {
				cufftSafeCall(hipfftExecC2C(plan, &deviceDataFile2[k*(numofFFTs * fftsize)], &deviceDataFile2[k*(numofFFTs * fftsize)], HIPFFT_FORWARD));
				hipDeviceSynchronize();
			}
		}
		
		cufftSafeCall(hipfftDestroy(plan));
		auto fft_elapsed = chrono::high_resolution_clock::now() - fftbeg;

		//CHECK: FFT (only for printing fft)
		//CudaSafeCall(hipMemcpy(hostDataFile1, deviceDataFile1, sizeof(hipfftComplex)*samplesWithOverlap, hipMemcpyDeviceToHost));
		//hipDeviceSynchronize();
		//writedata(samplesWithOverlap, hostDataFile1, "results/fft.bin");

		//COMPLEX CONJUGATE AND MULTIPLICATION
		auto multbeg = std::chrono::high_resolution_clock::now();
		numBlocks = (samplesWithOverlap + blockSize - 1) / blockSize;
		multip << <numBlocks, blockSize >> > (samplesWithOverlap, deviceDataFile1, deviceDataFile2, fftsize,interferometic);
		CudaCheckError();
		hipDeviceSynchronize();
		auto mult_elapsed = chrono::high_resolution_clock::now() - multbeg;
		
		//CHECK: MULTIPLICATION (only for printing multiplication result)
		//CudaSafeCall(hipMemcpy(hostDataFile1, deviceDataFile1, sizeof(hipfftComplex)*samplesWithOverlap, hipMemcpyDeviceToHost));
		//hipDeviceSynchronize();
		//writedata(samplesWithOverlap, hostDataFile1, "mult.txt");
		
		//IFFT
		auto ifftbeg = std::chrono::high_resolution_clock::now();
		planifftFunction(fftsize, numofFFTs, 0, &plan);
		hipDeviceSynchronize();
		for (k = 0; k < ddmQuant; k++) {
			cufftSafeCall(hipfftExecC2C(plan, &deviceDataFile1[k*(numofFFTs * fftsize)], &deviceDataFile1[k*(numofFFTs * fftsize)], HIPFFT_BACKWARD));
			hipDeviceSynchronize();
		}
		cufftSafeCall(hipfftDestroy(plan));
		auto ifft_elapsed = chrono::high_resolution_clock::now() - ifftbeg;

		//SCALE (To take back original signal it has to be devided for the fftsize)
		/*auto scalebeg = std::chrono::high_resolution_clock::now();
		numBlocks = (samplesWithOverlap + blockSize - 1) / blockSize;		
		scale << <numBlocks, blockSize >> > (samplesWithOverlap, deviceDataFile1, fftsize);
		CudaCheckError();
		hipDeviceSynchronize();
		auto scale_elapsed = chrono::high_resolution_clock::now() - scalebeg;*/

		//CHECK: IFFT 
		//CudaSafeCall(hipMemcpy(hostDataFile1, deviceDataFile1, sizeof(hipfftComplex)*samplesWithOverlap, hipMemcpyDeviceToHost)); 
		//hipDeviceSynchronize();	
		//writedata(samplesWithOverlap, hostDataFile1,  "results/IFFT.bin");

		//INCOHERENT SUM
		auto incohbeg = std::chrono::high_resolution_clock::now();
		numBlocks = (inchoerentNumofFFT*fftsize + blockSize - 1) / blockSize;
		inchoerentSum << <numBlocks, blockSize >> > (inchoerentNumofFFT*fftsize, deviceDataFile1, deviceIncoherentSum, quantofAverageIncoherent, fftsize);
		CudaCheckError(); 
		hipDeviceSynchronize();
		auto incho_elapsed = chrono::high_resolution_clock::now() - incohbeg;
		
		//CHECK: INCOHERENT
		//CudaSafeCall(hipMemcpy(hostDataFile1, deviceIncoherentSum, sizeof(Npp32f)*inchoerentNumofFFT*fftsize, hipMemcpyDeviceToHost));
		//hipDeviceSynchronize();
		//writeIncoh(inchoerentNumofFFT*fftsize, hostDataFile1, "results/incoh.bin");
		
		//MAXIMUM
		auto maxbeg = std::chrono::high_resolution_clock::now();
		maxCompute(inchoerentNumofFFT, deviceIncoherentSum, fftsize, devicearrayMaxs, devicearrayPos, pMaxDeviceBuffer,samplesAvoidMax);
		hipDeviceSynchronize();
		CudaSafeCall(hipMemcpy(hostarrayPos, devicearrayPos, sizeof(int)*inchoerentNumofFFT, hipMemcpyDeviceToHost));
		CudaSafeCall(hipMemcpy(hostarrayMaxs, devicearrayMaxs, sizeof(Npp32f)*inchoerentNumofFFT, hipMemcpyDeviceToHost));
		hipDeviceSynchronize();
		auto max_elapsed = chrono::high_resolution_clock::now() - maxbeg;

		//SAVE PEAKS
		auto savepbeg = std::chrono::high_resolution_clock::now();
		if (ddmQuant > 1) {
			//numBlocks = ((numofFFTs / quantofAverageIncoherent) + blockSize - 1) / blockSize;
			selectMaxs << <1, blockSize >> > (numofFFTs, quantofAverageIncoherent, ddmQuant, devicearrayPos, devicearrayMaxs);
			CudaCheckError();
			hipDeviceSynchronize();

			//CHECK MAX
			//CudaSafeCall(hipMemcpy(&checkMax, devicearrayPos, sizeof(int)*inchoerentNumofFFT, hipMemcpyDeviceToHost));
			//cout << checkMax[0] << "\n";
		}
		numBlocks = (numofFFTs*peakSamplesToSave*ddmQuant + blockSize - 1) / blockSize;
		savePeak << <numBlocks, blockSize >> > (numofFFTs, deviceDataFile1, deviceDataToSave, peakSamplesToSave, quantofAverageIncoherent, fftsize, devicearrayPos,ddmQuant);
		CudaCheckError();
		hipDeviceSynchronize();
		CudaSafeCall(hipMemcpy(hostDataFile1, deviceDataToSave, sizeof(hipComplex)*numofFFTs*peakSamplesToSave*ddmQuant, hipMemcpyDeviceToHost));
		auto savep_elapsed = chrono::high_resolution_clock::now() - savepbeg;
		
		//STD
		auto stdbeg = std::chrono::high_resolution_clock::now();
		stdCompute(inchoerentNumofFFT, deviceIncoherentSum, fftsize, devicearrayStd, hostarrayPos, pStdDeviceBuffer, peakRangeStd,stdLength, devicearrayMean);
		hipDeviceSynchronize();
		CudaSafeCall(hipMemcpy(hostarrayStd, devicearrayStd, sizeof(Npp32f)*inchoerentNumofFFT, hipMemcpyDeviceToHost));
		CudaSafeCall(hipMemcpy(hostarrayMean, devicearrayMean, sizeof(Npp32f)*inchoerentNumofFFT, hipMemcpyDeviceToHost));
		hipDeviceSynchronize();
		auto std_elapsed = chrono::high_resolution_clock::now() - stdbeg;

		//OUTPUT
		auto writeBeg = chrono::high_resolution_clock::now();
		outputName = resultDirectory+"Maximums" + to_string(i) + ".bin";
		writeMaxs(inchoerentNumofFFT, hostarrayMaxs, hostarrayPos, hostarrayStd, hostarrayMean,doppler[i], outputName,i, ddmRes,ddmQuant,numofFFTs / quantofAverageIncoherent);
		
		outputName = resultDirectory + "PeaksIteration" + to_string(i) + ".bin";
		cout << outputName << "\n";
		writedata(numofFFTs*peakSamplesToSave*ddmQuant, hostDataFile1, outputName);
	
		//ELAPSED TIME
		auto elapsed_write = chrono::high_resolution_clock::now() - writeBeg;
		auto elapsed_total = chrono::high_resolution_clock::now() - begin;

		read_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(elapsed_read).count();
		write_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(elapsed_write).count();
		elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(elapsed_total).count();
		mask_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(mask_elapsed).count();
		doppler_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(doppler_elapsed).count();
		fft_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(fft_elapsed).count();
		mult_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(mult_elapsed).count();
		ifft_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(ifft_elapsed).count();
		extenddop_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(extenddop_elapsed).count();
		incho_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(incho_elapsed).count();
		max_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(max_elapsed).count();
		savep_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(savep_elapsed).count();
		std_elapsed_secs[i] = chrono::duration_cast<chrono::microseconds>(std_elapsed).count();
	}
	outputName = resultDirectory + "Times.txt";
	writetime(numofDataLines, outputName, read_elapsed_secs, write_elapsed_secs, elapsed_secs,
		mask_elapsed_secs, extenddop_elapsed_secs, doppler_elapsed_secs,
		fft_elapsed_secs, mult_elapsed_secs, ifft_elapsed_secs, incho_elapsed_secs
		, max_elapsed_secs, savep_elapsed_secs, std_elapsed_secs);
	
	//FREE MEMORY
	//cufftSafeCall(hipfftDestroy(plan));
	//cufftSafeCall(hipfftDestroy(inverseplan));
	hipFree(deviceDataFile1);
	hipFree(deviceDataFile2);
	hipFree(deviceIncoherentSum);
	hipFree(devicearrayPos);
	hipFree(deviceBytesOfData);
	hipFree(devicearrayMaxs);
	hipFree(devicearrayMean);
	hipFree(deviceDataToSave);
	hipFree(pStdDeviceBuffer);
	hipFree(pMaxDeviceBuffer);
	hipFree(devicearrayStd);
	hipDeviceReset();
	delete[] fileDataNames;
	if (interferometic == true) {
		delete[] fileRefName;
		delete[] dataOffsetBegInterferometric;
	}
	delete[] hostBytesOfData;
	delete[] hostarrayPos;
	delete[] hostarrayMaxs;
	delete[] hostarrayMean;
	delete[] hostarrayStd;
	delete[] hostDataFile2;
	delete[] hostDataFile1;
	delete[] dataOffsetBeg;
	delete[] dataOffsetEnd;
	delete[] doppler;
	delete[] read_elapsed_secs;
	delete[] write_elapsed_secs;
	delete[] elapsed_secs;
	delete[] mask_elapsed_secs;
	delete[] doppler_elapsed_secs ;
	delete[] mult_elapsed_secs ;
	delete[] fft_elapsed_secs;
	delete[] ifft_elapsed_secs;
	delete[] extenddop_elapsed_secs;
	delete[] incho_elapsed_secs ;
	delete[] max_elapsed_secs ;
	delete[] savep_elapsed_secs ;
	delete[] std_elapsed_secs;
	return 0;
}
