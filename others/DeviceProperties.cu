//==========================================================================
// Author: Oriol Cervelló (oriol.cn [at] protonmail.com) 
//==========================================================================
// License: GNU GPLv3.0
// Copyright (C) 2019  Oriol Cervelló
//
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
// 
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
// 
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.
//==========================================================================
#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
using namespace std;



int main() {
	//-------
	//DEVICE MANAGEMENT

	hipDeviceReset();
	int count;
	hipGetDeviceCount(&count);
	fprintf(stderr, "count devices: %i \n", count);
	for (int i = 0; i < count; i++) {
		hipDeviceProp_t aa;
		hipSetDevice(i);
		hipGetDeviceProperties(&aa, i);

		
		
		fprintf(stderr, "Device %i ", i);
		fprintf(stderr, ":\n  Name: %s \n", aa.name);
		fprintf(stderr, "  maxThreadsPerBlock: %i \n", aa.maxThreadsPerBlock);
		fprintf(stderr, "  max dim of block of x: %i \n", aa.maxThreadsDim[0]);
		fprintf(stderr, "  max dim of block of y: %i \n", aa.maxThreadsDim[1]);
		
		
		size_t freeMem, totalMem;

		hipMemGetInfo(&freeMem, &totalMem);

		fprintf(stderr, "  Memory: \n");
		fprintf(stderr, "   Free = %zu, Total = %zu\n", freeMem, totalMem);
	}
	
}
